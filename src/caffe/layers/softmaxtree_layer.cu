#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layers/softmaxtree_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

dim3 caffe_gridsize(uint32_t n) {
    uint32_t x = (n + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;;
    uint32_t y = 1;
    if (x > 65535) {
        x = ceil(sqrt(x));
        y = (n + x * CAFFE_CUDA_NUM_THREADS - 1) / (x * CAFFE_CUDA_NUM_THREADS);
    }
    dim3 d(x, y, 1);

    return d;
}

template <typename Dtype>
__global__ void kernel_subtract_max(const int num, const int channels, const int spatial_dim, const int groups,
                                    const int* group_offset_data, const int* group_size_data, Dtype* data) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= num * groups * spatial_dim)
        return;
    int s = index % spatial_dim;
    index /= spatial_dim;
    int g = index % groups;
    int n = index / groups;
    auto offset = group_offset_data[g];
    auto size = group_size_data[g];
    Dtype maxval = -FLT_MAX;
    for (int j = 0; j < size; ++j) {
        if (data[(n * channels + offset + j) * spatial_dim + s] > maxval)
            maxval = data[(n * channels + offset + j) * spatial_dim + s];
    }
    // TODO: Use dynamic parallelism for devices with 3.5 compute capability
    // Subtract the max
    for (int j = 0; j < size; ++j)
        data[(n * channels + offset + j) * spatial_dim + s] -= maxval;
}

template <typename Dtype>
__global__ void kernel_exp(const int count, Dtype* data) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= count)
        return;
    data[index] = exp(data[index]);
}

template <typename Dtype>
__global__ void kernel_div_sum(const int num, const int channels, const int spatial_dim, const int groups,
                               const int* group_offset_data, const int* group_size_data, Dtype* data) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= num * groups * spatial_dim)
        return;
    int s = index % spatial_dim;
    index /= spatial_dim;
    int g = index % groups;
    int n = index / groups;
    auto offset = group_offset_data[g];
    auto size = group_size_data[g];
    Dtype sum = 0;
    for (int j = 0; j < size; ++j)
      sum += data[(n * channels + offset + j) * spatial_dim + s];
    // TODO: Use dynamic parallelism for devices with 3.5 compute capability
    // divide by sum
    for (int j = 0; j < size; ++j)
        data[(n * channels + offset + j) * spatial_dim + s] /= sum;
}

template <typename Dtype>
__global__ void kernel_subtract_dot(const int num, const int channels, const int spatial_dim, const int groups,
                                    const int* group_offset_data, const int* group_size_data, 
                                    const Dtype* data_1, const Dtype* data_2, Dtype* out) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= num * groups * spatial_dim)
        return;
    int s = index % spatial_dim;
    index /= spatial_dim;
    int g = index % groups;
    int n = index / groups;
    auto offset = group_offset_data[g];
    auto size = group_size_data[g];
    Dtype dot = 0;
    for (int j = 0; j < size; ++j) {
        dot += (data_1[(n * channels + offset + j) * spatial_dim + s]
                * data_2[(n * channels + offset + j) * spatial_dim + s]);
    }
    // TODO: Use dynamic parallelism for devices with 3.5 compute capability
    // subtract the dot
    for (int j = 0; j < size; ++j)
        out[(n * channels + offset + j) * spatial_dim + s] -= dot;
}

template <typename Dtype>
void SoftmaxTreeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  auto group_offset_data = (const int*)softmax_tree_.group_offset->gpu_data();
  auto group_size_data = (const int*)softmax_tree_.group_size->gpu_data();
  const auto groups = softmax_tree_.groups;
  int count = bottom[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, bottom_data, top_data);
  // We need to subtract the per-group max to avoid numerical issues, compute the exp,
  // and then per-group normalize.
  kernel_subtract_max<Dtype><<<caffe_gridsize(outer_num_ * groups * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, groups,
                                group_offset_data, group_size_data, top_data);
  // exponentiate
  kernel_exp<Dtype><<<caffe_gridsize(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data);
  // per-group sum after exp, and divide
  kernel_div_sum<Dtype><<<caffe_gridsize(outer_num_ * groups * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, groups, 
                                group_offset_data, group_size_data, top_data);
}

template <typename Dtype>
void SoftmaxTreeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  auto group_offset_data = (const int*)softmax_tree_.group_offset->gpu_data();
  auto group_size_data =(const int*)softmax_tree_.group_size->gpu_data();
  const auto groups = softmax_tree_.groups;
  int count = top[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, top_diff, bottom_diff);
  // Compute per-group inner1d(top_diff, top_data) and subtract them from the bottom diff.
  kernel_subtract_dot<Dtype><<<caffe_gridsize(outer_num_ * groups * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, groups,
                                group_offset_data, group_size_data, 
                                top_diff, top_data, bottom_diff);
  // elementwise multiplication
  caffe_gpu_mul<Dtype>(count, bottom_diff, top_data, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxTreeLayer);


}  // namespace caffe
