#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hip/hip_runtime.h"

#include "caffe/layers/yolo_bbs_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_yolo_bbs(int batches, int num_anchor, int height, int width,
                                const Dtype* blob_xy_data, const Dtype* blob_wh_data,
                                const Dtype* biases_data,
                                Dtype* bbs_data) {
    CUDA_KERNEL_LOOP(index, batches * num_anchor * height * width) {
        int b = index / (num_anchor * height * width);
        int left = index % (num_anchor * height * width);
        int n = left / (height * width);
        left = left % (height * width);
        int j = left / width;
        int i = left % width;
        Dtype* curr_bbs_data = bbs_data + index * 4;
        int offset_double_bnji = b * (2 * num_anchor) * height * width + n * height * width + j * width + i;
        int offset_double_bnji_next = offset_double_bnji + num_anchor * height * width;
        *(curr_bbs_data + 0) = (*(blob_xy_data + offset_double_bnji) + i) / width;
        *(curr_bbs_data + 1) = (*(blob_xy_data + offset_double_bnji_next) + j) / height;
        double w = *(blob_wh_data + offset_double_bnji);
        double h = *(blob_wh_data + offset_double_bnji_next);
        *(curr_bbs_data + 2) = exp(w) * biases_data[2 * n] / width;
        *(curr_bbs_data + 3) = exp(h) * biases_data[2 * n + 1] / height;
    }
}

template <typename Dtype>
__global__ void kernel_correct_bbs(int total, 
                                   int im_w, int im_h, int netw, int neth,
                                   int new_w, int new_h,
                                   Dtype* bbs_data) {
    CUDA_KERNEL_LOOP(i, total) {
        Dtype x = bbs_data[4 * i + 0];
        Dtype y = bbs_data[4 * i + 1];
        Dtype w = bbs_data[4 * i + 2];
        Dtype h = bbs_data[4 * i + 3];

        x = (x - (netw - new_w) / 2. / netw) / ((Dtype)new_w / netw);
        y = (y - (neth - new_h) / 2. / neth) / ((Dtype)new_h / neth);
        w *= (Dtype)netw / new_w;
        h *= (Dtype)neth / new_h;
        x *= im_w;
        w *= im_w;
        y *= im_h;
        h *= im_h;
        bbs_data[4 * i + 0] = x;
        bbs_data[4 * i + 1] = y;
        bbs_data[4 * i + 2] = w;
        bbs_data[4 * i + 3] = h;
    }
}

template <typename Dtype>
void YoloBBsLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    int blob_idx = 0;
    auto blob_xy = bottom[blob_idx++];
    auto blob_wh = bottom[blob_idx++];

    auto bbs = top[0];

    int batches = blob_xy->num();
    int height = blob_xy->height();
    int width = blob_xy->width();
    int num_anchor = blob_xy->channels() / 2;

    kernel_yolo_bbs<Dtype> << <CAFFE_GET_BLOCKS(batches * num_anchor * height * width), CAFFE_CUDA_NUM_THREADS >> >(
        batches, num_anchor, height, width,
        blob_xy->gpu_data(), blob_wh->gpu_data(), 
        biases_.gpu_data(),
        bbs->mutable_gpu_data());
    CUDA_POST_KERNEL_CHECK;

    if (bottom.size() < 3)
        return;

    auto blob_imageinfo = bottom[2];

    int net_h = feat_stride_ * height;
    int net_w = feat_stride_ * width;

    auto im_info = blob_imageinfo->cpu_data();
    int im_h = im_info[0];
    int im_w = im_info[1];
    // when used for Caffe timing, im_w and im_h might be 0 and we need to give them valid values.
    if (im_w == 0)
        im_w = net_w;
    if (im_h == 0)
        im_h = net_h;

    int new_w = 0;
    int new_h = 0;
    if (((Dtype)net_w / im_w) < ((Dtype)net_h / im_h)) {
        new_w = net_w;
        new_h = (im_h * net_w) / im_w;
    } else {
        new_h = net_h;
        new_w = (im_w * net_h) / im_h;
    }

    kernel_correct_bbs<Dtype> << <CAFFE_GET_BLOCKS(bbs->count() / 4), CAFFE_CUDA_NUM_THREADS >> > (
        bbs->count() / 4,
        im_w, im_h, net_w, net_h,
        new_w, new_h,
        bbs->mutable_gpu_data());
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(YoloBBsLayer);

}  // namespace caffe
